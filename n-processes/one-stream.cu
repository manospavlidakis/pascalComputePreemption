#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// std::system includes
#include <cstdio>

// CUDA-C includes
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <helper_string.h>


// throw error on equality
#define ERR_EQ(X,Y) do { if ((X) == (Y)) { \
  fprintf(stderr,"Error in %s at %s:%d\n",__func__,__FILE__,__LINE__); \
  exit(-1);}} while(0)

// throw error on difference
#define ERR_NE(X,Y) do { if ((X) != (Y)) { \
  fprintf(stderr,"Error in %s at %s:%d\n",__func__,__FILE__,__LINE__); \
  exit(-1);}} while(0)

#define ROUND_UP(N, BASE) \
  (N + BASE - 1) / BASE

// copy from source -> destination arrays
__global__ void slow_kernel(int *dst, int *src, int n) {
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  #define DELAY 2048
  for (volatile int i = 0; i < DELAY; i++)
    ;
  if (id < n) {
    dst[id] = src[id];
  }
}

__global__ void fast_kernel(int *dst, int *src, int n) {
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id < n) {
    dst[id] = src[id];
  }
}

// initialize memory
void mem_init(int *buf, size_t n) {
  for (int i = 0; i < n; i++) {
    buf[i] = i;
  }
}

// Forward declarations
hipError_t setup_memory(int* src[], int* dst[], size_t size, size_t n_regions);
hipError_t iterate_stream(bool delay, size_t iterations, int* src[],
    int* dst[], size_t n_regions);

int main(int argc, char **argv) {
  hipDeviceProp_t device_prop;
  int dev_id;

  printf("Starting [%s]...\n", argv[0]);

  // set device
  dev_id = findCudaDevice(argc, (const char **) argv);
  checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));
  if ((device_prop.major << 4) + device_prop.minor < 0x35) {
    fprintf(stderr,
        "%s requires Compute Capability of SM 3.5 or higher to run.\nexiting...\n",
        argv[0]);
    exit (EXIT_WAIVED);
  }

  // command line args
  const bool delay = checkCmdLineFlag(argc, (const char **) argv, "delay");
  const size_t size = getCmdLineArgumentInt(argc, (const char **) argv, "size");
  const int priority = getCmdLineArgumentInt(argc, (const char **) argv, "priority");
  const int iterations = getCmdLineArgumentInt(argc, (const char **) argv, "iterations");

  // get the range of priorities available
  // [ greatest_priority, least_priority ]
  int priority_low;
  int priority_hi;
  checkCudaErrors(hipDeviceGetStreamPriorityRange(&priority_low,
      &priority_hi));

  if (size == 0 || iterations == 0) {
    fprintf(stderr,
        "Please provide --size=<int> --priority=<int> --iterations=<int> and --delay(optional) flags.\nexting...\n");
    exit (EXIT_FAILURE);
  }
  
  if (priority_hi > priority || priority_low < priority) {
    fprintf(stderr, "Priority must be within %d and %d.\nexting...\n",
        priority_hi, priority_low);
    exit (EXIT_FAILURE);
  }

  // Create memory regions
#define N_MEMORY_REGIONS 8
  size_t n_regions = max(min(N_MEMORY_REGIONS, iterations / 4), 1);
  int *src[n_regions];
  int *dst[n_regions];
  
  setup_memory(src, dst, size, n_regions);

  hipDeviceSynchronize();
  // Create stream
  hipStream_t stream;
  checkCudaErrors(hipStreamCreateWithPriority(&stream,
      hipStreamNonBlocking, priority));
 
  // Compute number of threads and blocks
  int blockSize;
  int minGridSize;
  int gridSize;
  if (delay) {
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, slow_kernel);
  } else {
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, fast_kernel);    
  }
  gridSize = (size + blockSize - 1) / blockSize;
 
  // launch the kernel iteration times.
  // each consecutive launch uses a different memory region
  for (int i = 0; i < ROUND_UP(iterations, n_regions); i++) {
    for (int j = 0; j < n_regions; j++) {
      if (delay) {
        slow_kernel<<<gridSize, blockSize, 0, stream>>>(dst[j], src[j], size);
      } else {
        fast_kernel<<<gridSize, blockSize, 0, stream>>>(dst[j], src[j], size);
      }
    }
  }

  checkCudaErrors(hipStreamSynchronize(stream));

  // check results of the last computation
  for (int i = 0; i < n_regions; i++) {
    ERR_NE(memcmp(dst[i], src[i], size), 0);  
  }
  
  // Clean up
  for (int i = 0; i < n_regions; i++) {
    checkCudaErrors (hipFree(src[i]));
    checkCudaErrors(hipFree(dst[i]));
  }
  exit (EXIT_SUCCESS);
}

hipError_t setup_memory(int* src[], int* dst[], size_t size,
    size_t n_regions) {
  for (int i = 0; i < n_regions; i++) {
    checkCudaErrors(hipMallocManaged(&src[i], size * sizeof(int)));
    checkCudaErrors(hipMallocManaged(&dst[i], size * sizeof(int)));
    mem_init(src[i], size);
    memset(dst[i], 0, size);
  }
  return hipSuccess;
}

/* */
