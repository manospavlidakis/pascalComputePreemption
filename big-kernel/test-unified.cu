#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// std::system includes
#include <cstdio>

// CUDA-C includes
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <helper_string.h>

// Semaphore include
#include <sys/types.h>
#include <unistd.h>

// throw error on equality
#define ERR_EQ(X,Y) do { if ((X) == (Y)) { \
  fprintf(stderr,"Error in %s at %s:%d\n",__func__,__FILE__,__LINE__); \
  exit(-1);}} while(0)

// throw error on difference
#define ERR_NE(X,Y) do { if ((X) != (Y)) { \
  fprintf(stderr,"Error in %s at %s:%d\n",__func__,__FILE__,__LINE__); \
  exit(-1);}} while(0)

#define ROUND_UP(N, BASE) \
  (N + BASE - 1) / BASE

// copy from source -> destination arrays
__device__ void slow_kernel(int *dst, int *src, int n, int delay) {
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  for (volatile int i = 0; i < delay; i++);
  if (id < n) {
    dst[id] = src[id];
  }
}

// Named kernels for easier profiling
__global__ void low_priority(int *dst, int *src, int n, int delay) {
  slow_kernel(dst, src, n, delay);
}

__global__ void high_priority(int *dst, int *src, int n, int delay) {
  slow_kernel(dst, src, n, delay);
}

// initialize memory
void mem_init(int *buf, size_t n) {
  for (int i = 0; i < n; i++) {
    buf[i] = i;
  }
}

// Forward declarations
hipError_t setup_memory(int* src[], int* dst[], size_t size, size_t n_regions);
void run_experiment(const int priority, const int size, const int iterations,
    const int delay);

int main(int argc, char **argv) {
  hipDeviceProp_t device_prop;
  int dev_id;

  fprintf(stderr, "Starting [%s]...\n", argv[0]);

  // set device
  dev_id = findCudaDevice(argc, (const char **) argv);
  checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));
  if ((device_prop.major << 4) + device_prop.minor < 0x35) {
    fprintf(stderr,
        "%s requires Compute Capability of SM 3.5 or higher to run.\nexiting...\n",
        argv[0]);
    exit (EXIT_WAIVED);
  }

  // command line args
  const int delay = getCmdLineArgumentInt(argc, (const char **) argv, "delay");
  const size_t size = getCmdLineArgumentInt(argc, (const char **) argv, "size");
  const int priority = getCmdLineArgumentInt(argc, (const char **) argv, "priority");
  const int iterations = getCmdLineArgumentInt(argc, (const char **) argv, "iterations");

  // get the range of priorities available
  // [ greatest_priority, least_priority ]
  int priority_low;
  int priority_hi;
  checkCudaErrors(hipDeviceGetStreamPriorityRange(&priority_low,
      &priority_hi));

  if (size == 0 || iterations == 0) {
    fprintf(stderr,
        "Please provide --size=<int> --priority=<int> --iterations=<int> "
        "and --delay=<int> (optional) flags.\nexting...\n");
    exit (EXIT_FAILURE);
  } else {
    fprintf(stderr, "Called with arguments size %zu, priority %d, iterations %d, and delay %d\n",
        size, priority, iterations, delay);
  }
  
  if (priority_hi > priority || priority_low < priority) {
    fprintf(stderr, "Priority must be within %d and %d.\nexting...\n",
        priority_hi, priority_low);
    exit (EXIT_FAILURE);
  }

  // Set kernel to run
  void (*kernel)(int*, int*, int, int) = priority ? &high_priority : &low_priority;
  
  // Create memory regions
#define N_MEMORY_REGIONS 8
  size_t n_regions = N_MEMORY_REGIONS;
  int *src[n_regions];
  int *dst[n_regions];
  
  setup_memory(src, dst, size, n_regions);
  hipDeviceSynchronize();

  // Create stream
  hipStream_t stream;
  checkCudaErrors(hipStreamCreateWithPriority(&stream,
      hipStreamNonBlocking, priority));
 
  // Compute number of threads and blocks
  int blockSize;
  int minGridSize;
  int gridSize;
  hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, kernel);
  gridSize = (size + blockSize - 1) / blockSize;
 
  // launch the kernel iteration times.
  // each consecutive launch uses a different memory region
  hipEvent_t start, end;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&end));
  checkCudaErrors(hipEventRecord(start, stream));
  for (int i = 0; i < iterations; i++) {
    for (int j = 0; j < n_regions; j++, i++) {
      if (i >= iterations) {
        break;
      }
      kernel<<<gridSize, blockSize, 0, stream>>>(dst[j], src[j], size,
          delay);
      checkCudaErrors(hipStreamSynchronize(stream));
    }
  }
  checkCudaErrors(hipEventRecord(end, stream));
  checkCudaErrors(hipEventSynchronize(end));

  // check results of the last computation
  for (int i = 0; i < n_regions && i < iterations; i++) {
    ERR_NE(memcmp(dst[i], src[i], size), 0);  
  }
  
  // Clean up
  for (int i = 0; i < n_regions; i++) {
    checkCudaErrors (hipFree(src[i]));
    checkCudaErrors(hipFree(dst[i]));
  }
  // Print out average time
  float ms;
  checkCudaErrors(hipEventElapsedTime(&ms, start, end));

  // size iterations ms average
  printf("%zu, %d, %f, %f\n", size, iterations, ms, ms / (float) iterations);
  exit (EXIT_SUCCESS);
}

hipError_t setup_memory(int* src[], int* dst[], size_t size,
    size_t n_regions) {
  for (int i = 0; i < n_regions; i++) {
    checkCudaErrors(hipMallocManaged(&src[i], size * sizeof(int)));
    checkCudaErrors(hipMallocManaged(&dst[i], size * sizeof(int)));
    mem_init(src[i], size);
    memset(dst[i], 0, size);
  }
  return hipSuccess;
}

