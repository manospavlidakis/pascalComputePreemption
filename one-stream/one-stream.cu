#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// std::system includes
#include <cstdio>

// CUDA-C includes
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>

#define TOTAL_SIZE  256*1024*1024

// # threadblocks
#define TBLOCKS 1024
#define THREADS  512

// throw error on equality
#define ERR_EQ(X,Y) do { if ((X) == (Y)) { \
  fprintf(stderr,"Error in %s at %s:%d\n",__func__,__FILE__,__LINE__); \
  exit(-1);}} while(0)

// throw error on difference
#define ERR_NE(X,Y) do { if ((X) != (Y)) { \
  fprintf(stderr,"Error in %s at %s:%d\n",__func__,__FILE__,__LINE__); \
  exit(-1);}} while(0)

// copy from source -> destination arrays
__global__ void slow_kernel(int *dst, int *src, size_t n)
{
  int num = gridDim.x * blockDim.x;
  int id = blockDim.x * blockIdx.x + threadIdx.x;

  int ret = 0; // accumulate garbage computation in ret and write it to dest[i]
  for (int i = id; i < n / sizeof(int); i += num) {
    for (int j = 0; j < 1000; j++) {
      ret += src[i] * j;
    }
    dst[i] = ret;
  }
}

// initialise memory
void mem_init(int *buf, size_t n) {
  for (int i = 0; i < n / sizeof(int); i++) {
    buf[i] = i;
  }
}

// Forward declaration
int preempt_stream(int, int);

int main(int argc, char **argv)
{
  hipDeviceProp_t device_prop;
  int dev_id;

  printf("Starting [%s]...\n", argv[0]);

  // set device
  dev_id = findCudaDevice(argc, (const char **) argv);
  checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));

  if ((device_prop.major << 4) + device_prop.minor < 0x35)
  {
    fprintf(stderr, "%s requires Compute Capability of SM 3.5 or higher to run.\nexiting...\n", argv[0]);
    exit(EXIT_WAIVED);
  }

  // get the range of priorities available
  // [ greatest_priority, lowest_priority ]
  int priority_low;
  int priority_hi;
  checkCudaErrors(hipDeviceGetStreamPriorityRange(&priority_low, &priority_hi));
  printf("CUDA stream priority range: LOW: %d to HIGH: %d\n", priority_low, priority_hi);

  preempt_stream(priority_low, priority_hi);

  exit(EXIT_SUCCESS);
}

/**
 * Creates a stream with low priority and starts a long-running kernel on it.
 * Creates a stream with high priority and runs a short-running kernel on it,
 * after the low-priority kernel has begun.
 * -- If preemption works, the run time of the low priority kernel should
 *    be extended by the runtime of the high priority kernel which preempts it.
 */
int preempt_stream(int priority_low, int priority_hi) {
  // Create streams
  size_t n_streams = 2; // Two streams (low and high)
                        // let index 0 hold low and 1 hold high
  hipStream_t streams[n_streams];
  checkCudaErrors(hipStreamCreateWithPriority(&streams[0],
        hipStreamNonBlocking, priority_low));
  checkCudaErrors(hipStreamCreateWithPriority(&streams[1],
        hipStreamNonBlocking, priority_hi));

  size_t size;
  size = TOTAL_SIZE;

  // initialise host data
  int *h_src[n_streams];
  for (int i = 0; i < n_streams; i++) {
    ERR_EQ(h_src[i] = (int *) malloc(size), NULL);
    mem_init(h_src[i], size);
  }

  // initialise device data
  int *h_dst[n_streams];
  for (int i = 0; i < n_streams; i++) {
    ERR_EQ(h_dst[i] = (int *) malloc(size), NULL);
    memset(h_dst[i], 0, size);
  }

  // copy source data -> device
  int *d_src[n_streams];
  for (int i = 0; i < n_streams; i++) {
    checkCudaErrors(hipMalloc(&d_src[i], size));
    checkCudaErrors(hipMemcpy(d_src[i], h_src[i], size, hipMemcpyHostToDevice));
  }

  // allocate memory for memcopy destination
  int *d_dst[n_streams];
  for (int i = 0; i < n_streams; i++) {
    checkCudaErrors(hipMalloc(&d_dst[i], size));
  }

  /* */

  // Begin profilling
  checkCudaErrors(hipProfilerStart());

  // Time low priority on its own
  {
    hipEvent_t start, end;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&end));
    checkCudaErrors(hipEventRecord(start, streams[0]));
    slow_kernel<<<TBLOCKS, THREADS, 0, streams[0]>>>(d_dst[0], d_src[0], TOTAL_SIZE);
    checkCudaErrors(hipEventRecord(end, streams[0]));
    checkCudaErrors(hipEventSynchronize(end));

    float ms;
    checkCudaErrors(hipEventElapsedTime(&ms, start, end));
    printf("Low priority solo elapsed time %0.6f ms\n", ms);
  }

  // Time high priority on its own
  {
    hipEvent_t start, end;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&end));
    checkCudaErrors(hipEventRecord(start, streams[1]));
    slow_kernel<<<TBLOCKS, THREADS, 0, streams[1]>>>(d_dst[1], d_src[1], TOTAL_SIZE);
    checkCudaErrors(hipEventRecord(end, streams[1]));
    checkCudaErrors(hipEventSynchronize(end));

    float ms;
    checkCudaErrors(hipEventElapsedTime(&ms, start, end));
    printf("High priority solo elapsed time %0.6f ms\n", ms);
  }

  // Start low priority then interrupt it with high priority
  {
    // create some events
    hipEvent_t ev_start[n_streams];
    hipEvent_t ev_end[n_streams];
    for (int i = 0; i < n_streams; i++) {
      checkCudaErrors(hipEventCreate(&ev_start[i]));
      checkCudaErrors(hipEventCreate(&ev_end[i]));
    }
  
    checkCudaErrors(hipEventRecord(ev_start[0], streams[0]));
    slow_kernel<<<TBLOCKS, THREADS, 0, streams[0]>>>(d_dst[0], d_src[0], TOTAL_SIZE);
    checkCudaErrors(hipEventRecord(ev_end[0], streams[0]));

    // synchronize on the start, so we launch this after the low priority kernel has started

    checkCudaErrors(hipEventSynchronize(ev_start[0]));

    checkCudaErrors(hipEventRecord(ev_start[1], streams[1]));
    slow_kernel<<<TBLOCKS, THREADS, 0, streams[1]>>>(d_dst[1], d_src[1], TOTAL_SIZE);
    checkCudaErrors(hipEventRecord(ev_end[1], streams[1]));
    
    checkCudaErrors(hipEventSynchronize(ev_end[1]));
    checkCudaErrors(hipEventSynchronize(ev_end[0]));

    float ms[n_streams];
    for (int i = 0; i < n_streams; i++) {
      checkCudaErrors(hipEventElapsedTime(&ms[i], ev_start[i], ev_end[i]));
    }
    printf("Low priority preempted by high priority test\n");
    printf("Low priority elapsed time %0.6f ms\n", ms[0]);
    printf("High priority elapsed time %0.6f ms\n", ms[1]);

  }

  // Stop profiling
  checkCudaErrors(hipProfilerStop());

  /* */

  size = TOTAL_SIZE;
  for (int i = 0; i < n_streams; i++) {
    checkCudaErrors(hipMemcpy(h_dst[i], d_dst[i], size, hipMemcpyDeviceToHost));
  }

  // check results of kernels
  /*
  // If we were doing some easily checkable computation, we could 
  // verify that the result is correct here
  for (int i = 0; i < n_streams; i++) {
    ERR_NE(memcmp(h_dst[i], h_src[i], size), 0);
  }
  */

  // Clean up
  for (int i = 0; i < n_streams; i++) {
    checkCudaErrors(hipFree(d_src[i]));
    checkCudaErrors(hipFree(d_dst[i]));
  }

  return 0;
}
